#include "hip/hip_runtime.h"
//===------------------------------------------------------------*- C++ -*-===//
//
// Copyright 2018 Battelle Memorial Institute
//
//===----------------------------------------------------------------------===//

#include <hiprand/hiprand_kernel.h>
#include <chrono>
#include <cstring>
#include <iostream>
#include <unordered_map>

#include "im/cuda/cuda_generate_rrr_sets.h"
#include "im/cuda/cuda_graph.cuh"
#include "im/cuda/cuda_utils.h"

namespace im {

using mask_word_t = typename cuda_GraphTy::vertex_type;
constexpr size_t MAX_SET_SIZE = 32;

// tested configurations:
// + 1 walk per thread:
// - warp_step = 1
//
// + 1 walk per warp:
// - warp_step = cuda_prop.warpSize
//
// + 1 walk per block:
// - warp step = 1
// - block_size = 1
struct cuda_conf_t {
  hipDeviceProp_t cuda_prop;
  const cuda_GraphTy *graph = nullptr;

  // host-side buffers
  mask_word_t **res_masks = nullptr;
  size_t *active_batch_sizes = nullptr;

  // device-side buffers
  cuda_graph<cuda_GraphTy> *d_graph = nullptr;
  mask_word_t **d_res_masks = nullptr;
  hiprandState **d_rng_states = nullptr;

  // sizing
  size_t grid_size = 0, block_size = 0, n_blocks = 0;
  size_t warp_step = 0;   // 1: per-thread, warp-size: per-warp
  size_t batch_size = 0;  // walks per batch
  size_t mask_words = 0;

  // CUDA streams
  hipStream_t *streams = nullptr;
  size_t n_streams = 0;
} cuda_conf;

size_t next_stream_id(size_t s) { return (s + 1) % cuda_conf.n_streams; }
size_t prev_stream_id(size_t s) {
  return (s > 0 ? s : cuda_conf.n_streams) - 1;
}

__global__ void kernel_rng_setup(hiprandState *d_rng_states,
                                 unsigned long long seed, size_t base,
                                 size_t warp_size) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid % warp_size == 0) {
    int wid = tid / warp_size;
    hiprand_init(seed, base + wid, 0, d_rng_states + wid);
  }
}

#if CUDA_PROFILE
enum breakdown_tag { KERNEL, COPY, POSTPROC };
std::unordered_map<breakdown_tag, std::vector<std::chrono::nanoseconds>>
    profile_breakdown;

void print_profile(breakdown_tag tag, const std::string &label) {
  std::sort(profile_breakdown[tag].begin(), profile_breakdown[tag].end());
  auto &sample(profile_breakdown[tag]);
  std::chrono::microseconds tot{0};
  for (auto &x : sample)
    tot += std::chrono::duration_cast<std::chrono::microseconds>(x);
  std::cout << "*** tag: " << label << "\n*** "
            << "cnt=" << sample.size() << "\tmin=" << sample[0].count()
            << "\tmed=" << sample[sample.size() / 2].count()
            << "\tmax=" << sample.back().count() << "\ttot=" << tot.count()
            << "us\n";
}
#endif

void cuda_init(const cuda_GraphTy &G, unsigned long long seed,
               im::linear_threshold_tag &&model_tag) {
  hipError_t e;
  hipGetDeviceProperties(&cuda_conf.cuda_prop, 0);
  // copy graph to device
  cuda_conf.graph = &G;
  cuda_conf.d_graph = make_cuda_graph(G);

  // sizing
  // cuda_conf.warp_step = 1;  // per thread
  cuda_conf.warp_step = cuda_conf.cuda_prop.warpSize;  // per warp
  cuda_conf.block_size = cuda_conf.warp_step * (1 << 0);
  cuda_conf.n_blocks = 1 << 15;
  cuda_conf.grid_size = cuda_conf.n_blocks * cuda_conf.block_size;
  cuda_conf.batch_size = cuda_conf.grid_size / cuda_conf.warp_step;
  cuda_conf.mask_words = MAX_SET_SIZE;

  // print sizing info
  CUDA_LOG("> *** CUDA_BATCHED sizing ***\n");
  CUDA_LOG("block-size = %d\n", cuda_conf.block_size);
  CUDA_LOG("n. blocks  = %d\n", cuda_conf.n_blocks);
  CUDA_LOG("warp size  = %d\n", cuda_conf.cuda_prop.warpSize);
  CUDA_LOG("grid size  = %d\n", cuda_conf.grid_size);
  CUDA_LOG("batch size = %d\n", cuda_conf.batch_size);
  CUDA_LOG("g-mem size = %d\n",
           cuda_conf.grid_size * cuda_conf.mask_words * sizeof(mask_word_t));

  // init streams
  cuda_conf.n_streams = 2;
  cuda_conf.streams =
      (hipStream_t *)malloc(cuda_conf.n_streams * sizeof(hipStream_t));
  for (size_t i = 0; i < cuda_conf.n_streams; ++i)
    hipStreamCreate(&cuda_conf.streams[i]);

  // allocate host-side memory for result masks
  auto batch_mask_size =
      cuda_conf.batch_size * cuda_conf.mask_words * sizeof(mask_word_t);
  cuda_conf.res_masks =
      (mask_word_t **)malloc(cuda_conf.n_streams * sizeof(mask_word_t *));
  for (size_t i = 0; i < cuda_conf.n_streams; ++i)
    cuda_conf.res_masks[i] = (mask_word_t *)malloc(batch_mask_size);
  cuda_conf.active_batch_sizes =
      (size_t *)malloc(cuda_conf.n_streams * sizeof(size_t));

  // allocate device-side memory for results masks
  cuda_conf.d_res_masks =
      (mask_word_t **)malloc(cuda_conf.n_streams * sizeof(mask_word_t *));
  for (size_t i = 0; i < cuda_conf.n_streams; ++i) {
    e = hipMalloc(&cuda_conf.d_res_masks[i], batch_mask_size);
    cuda_check(e, __FILE__, __LINE__);
  }

  // init rng
  cuda_conf.d_rng_states =
      (hiprandState **)malloc(cuda_conf.n_streams * sizeof(hiprandState));
  for (size_t i = 0; i < cuda_conf.n_streams; ++i) {
    hipMalloc(&cuda_conf.d_rng_states[i],
               cuda_conf.batch_size * sizeof(hiprandState));
    cuda_check(e, __FILE__, __LINE__);

    kernel_rng_setup<<<cuda_conf.n_blocks, cuda_conf.block_size>>>(
        cuda_conf.d_rng_states[i], seed, i * cuda_conf.batch_size,
        cuda_conf.warp_step);
    cuda_check(__FILE__, __LINE__);
  }
}

void cuda_init(const cuda_GraphTy &G, unsigned long long seed,
               im::independent_cascade_tag &&) {}

void cuda_fini(im::linear_threshold_tag &&) {
// print profiling
#if CUDA_PROFILE
  printf("*** profiling summary (time unit: ns) ***\n");

  // print sizing info
  printf("> *** CUDA_BATCHED sizing ***\n");
  printf("block-size = %d\n", cuda_conf.block_size);
  printf("n. blocks  = %d\n", cuda_conf.n_blocks);
  printf("warp size  = %d\n", cuda_conf.cuda_prop.warpSize);
  printf("grid size  = %d\n", cuda_conf.grid_size);
  printf("batch size = %d\n", cuda_conf.batch_size);
  printf("g-mem size = %d\n",
         cuda_conf.grid_size * cuda_conf.mask_words * sizeof(mask_word_t));

  print_profile(breakdown_tag::KERNEL, "kernel");
  print_profile(breakdown_tag::COPY, "device-to-host copy");
  print_profile(breakdown_tag::POSTPROC, "post-processing");
#endif

  // finalize streams and free memory
  for (size_t i = 0; i < cuda_conf.n_streams; ++i) {
    hipStreamDestroy(cuda_conf.streams[i]);
    assert(cuda_conf.res_masks[i]);
    free(cuda_conf.res_masks[i]);
    assert(cuda_conf.active_batch_sizes[i]);
    assert(cuda_conf.d_res_masks[i]);
    hipFree(cuda_conf.d_res_masks[i]);
    assert(cuda_conf.d_rng_states[i]);
    hipFree(cuda_conf.d_rng_states[i]);
  }
  assert(cuda_conf.streams);
  free(cuda_conf.streams);
  assert(cuda_conf.res_masks);
  free(cuda_conf.res_masks);
  assert(cuda_conf.active_batch_sizes);
  free(cuda_conf.active_batch_sizes);
  assert(cuda_conf.d_res_masks);
  free(cuda_conf.d_res_masks);
  assert(cuda_conf.d_rng_states);
  free(cuda_conf.d_rng_states);

  // cleanup
  destroy_cuda_graph(cuda_conf.d_graph);
}

void cuda_fini(im::independent_cascade_tag &&) {}

template <typename HostGraphTy>
__global__ void kernel_lt_per_thread(
    size_t bs, typename HostGraphTy::DestinationTy **index, size_t num_nodes,
    size_t warp_size, hiprandState *d_rng_states, mask_word_t *d_res_masks) {
  using destination_type = typename HostGraphTy::DestinationTy;
  using vertex_type = typename HostGraphTy::vertex_type;

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid % warp_size == 0) {
    int wid = tid / warp_size;
    if (wid < bs) {
      size_t res_size = 0;

      // init res memory
      auto d_res_mask = d_res_masks + wid * MAX_SET_SIZE;
      memset(d_res_mask, 0, MAX_SET_SIZE * sizeof(mask_word_t));

      // cache rng state
      auto rng_state = d_rng_states + wid;

      // select source node
      vertex_type src = hiprand(rng_state) % num_nodes;
      d_res_mask[res_size++] = src;

      float threshold;
      destination_type *first, *last;
      vertex_type v;
      while (src != num_nodes) {
        // rng
        threshold = hiprand_uniform(rng_state);

        // scan neighbor list
        first = index[src];
        last = index[src + 1];
        src = num_nodes;
        for (; first != last; ++first) {
          threshold -= first->weight;
          if (threshold <= 0) {
            // found candidate vertex
            v = first->vertex;

            // insert if not visited
            size_t i = 0;
            while (i < res_size && d_res_mask[i] != v) ++i;
            if (i == res_size) {
              src = v;
              d_res_mask[res_size++] = v;
            }
            break;
          }
        }
      }

      // mark end-of-set
      if (res_size < MAX_SET_SIZE) d_res_mask[res_size] = num_nodes;
    }  // end if active warp
  }    // end if active thread-in-warp
}  // namespace im

void batch_kernel(size_t stream_id) {
  CUDA_LOG("> [batch_kernel] stream_id=%d size=%d\n", stream_id,
           cuda_conf.active_batch_sizes[stream_id]);

#if CUDA_PROFILE
  auto start = std::chrono::high_resolution_clock::now();
#endif

  kernel_lt_per_thread<cuda_GraphTy><<<cuda_conf.n_blocks, cuda_conf.block_size,
                                       0, cuda_conf.streams[stream_id]>>>(
      cuda_conf.active_batch_sizes[stream_id], cuda_conf.d_graph->d_index_,
      cuda_conf.graph->num_nodes(), cuda_conf.warp_step,
      cuda_conf.d_rng_states[stream_id], cuda_conf.d_res_masks[stream_id]);
  cuda_check(__FILE__, __LINE__);

#if CUDA_PROFILE
  hipDeviceSynchronize();
  auto elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(
      std::chrono::high_resolution_clock::now() - start);
  profile_breakdown[breakdown_tag::KERNEL].push_back(elapsed);
#endif
}

void batch_d2h(size_t stream_id) {
  CUDA_LOG("> [batch_d2h] stream_id=%d size=%d\n", stream_id,
           cuda_conf.active_batch_sizes[stream_id]);

#if CUDA_PROFILE
  auto start = std::chrono::high_resolution_clock::now();
#endif
  hipMemcpyAsync(cuda_conf.res_masks[stream_id],
                  cuda_conf.d_res_masks[stream_id],
                  cuda_conf.active_batch_sizes[stream_id] *
                      cuda_conf.mask_words * sizeof(mask_word_t),
                  hipMemcpyDeviceToHost, cuda_conf.streams[stream_id]);

#if CUDA_PROFILE
  auto elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(
      std::chrono::high_resolution_clock::now() - start);
  profile_breakdown[breakdown_tag::COPY].push_back(elapsed);
#endif
}

void batch_build(uint8_t stream_id, cuda_res_t &rrr_sets, size_t bf) {
  CUDA_LOG("> [batch_build] waiting stream_id=%d\n", stream_id);
  hipStreamSynchronize(cuda_conf.streams[stream_id]);

  CUDA_LOG("> [batch_build] stream_id=%d size=%d first=%d\n", stream_id,
           cuda_conf.active_batch_sizes[stream_id], bf);

#if CUDA_PROFILE
  auto start = std::chrono::high_resolution_clock::now();
#endif
  for (size_t i = 0; i < cuda_conf.active_batch_sizes[stream_id]; ++i) {
    auto &rrr_set = rrr_sets[bf + i];
    rrr_set.reserve(MAX_SET_SIZE);
    auto res_mask = cuda_conf.res_masks[stream_id] + (i * cuda_conf.mask_words);
    for (size_t j = 0; j < cuda_conf.mask_words &&
                       res_mask[j] != cuda_conf.graph->num_nodes();
         ++j) {
      rrr_set.push_back(res_mask[j]);
    }

#if CUDA_CHECK
    check_lt(rrr_set, *cuda_conf.graph, bf + i);
#endif

    if (rrr_set.size() == MAX_SET_SIZE) {
      fprintf(stderr, "> an RRR set hit the maximum size %d\n", MAX_SET_SIZE);
      exit(1);
    }
  }
#if CUDA_PROFILE
  auto elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(
      std::chrono::high_resolution_clock::now() - start);
  profile_breakdown[breakdown_tag::POSTPROC].push_back(elapsed);
#endif
}

cuda_res_t CudaGenerateRRRSets(size_t theta,
                               im::linear_threshold_tag &&model_tag) {
  CUDA_LOG("> *** CudaGenerateRRRSets theta=%d ***\n", theta);

  cuda_res_t rrr_sets(theta);

  auto remainder = rrr_sets.size();
  size_t batch_first = 0, stream_id = 0;

  cuda_conf.active_batch_sizes[stream_id] =
      std::min(remainder, cuda_conf.batch_size);

  // async execute+copy first batch
  batch_kernel(stream_id);
  batch_d2h(stream_id);
  remainder -= cuda_conf.active_batch_sizes[stream_id];

  while (remainder) {
    stream_id = next_stream_id(stream_id);
    cuda_conf.active_batch_sizes[stream_id] =
        std::min(remainder, cuda_conf.batch_size);

    // async execute+copy batch i
    batch_kernel(stream_id);
    batch_d2h(stream_id);
    remainder -= cuda_conf.active_batch_sizes[stream_id];

    // build sets for batch i-1 to sets
    auto prev = prev_stream_id(stream_id);
    batch_build(prev, rrr_sets, batch_first);
    batch_first += cuda_conf.active_batch_sizes[prev];
  }

  // build sets for last batch
  batch_build(stream_id, rrr_sets, batch_first);

  return rrr_sets;
}  // namespace im

cuda_res_t CudaGenerateRRRSets(size_t theta,
                               im::independent_cascade_tag &&model_tag) {
  assert(false);
  return cuda_res_t{};
}

}  // namespace im
