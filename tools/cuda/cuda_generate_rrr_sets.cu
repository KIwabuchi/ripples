#include "hip/hip_runtime.h"
//===------------------------------------------------------------*- C++ -*-===//
//
// Copyright 2018 Battelle Memorial Institute
//
//===----------------------------------------------------------------------===//

#include <hiprand/hiprand_kernel.h>
#include <chrono>
#include <cstring>
#include <iostream>
#include <unordered_map>

#include "im/cuda/cuda_generate_rrr_sets.h"
#include "im/cuda/cuda_graph.cuh"
#include "im/cuda/cuda_utils.h"

namespace im {

using mask_word_t = typename cuda_GraphTy::vertex_type;
constexpr size_t MAX_SET_SIZE = 64;

// tested configurations:
// + 1 walk per thread:
// - warp_step = 1
//
// + 1 walk per warp:
// - warp_step = cuda_prop.warpSize
//
// + 1 walk per block:
// - warp step = 1
// - block_size = 1
struct cuda_conf_t {
  hipDeviceProp_t cuda_prop;
  const cuda_GraphTy *graph = nullptr;

  // host-side buffers
  mask_word_t *res_masks = nullptr;

  // device-side buffers
  cuda_graph<cuda_GraphTy> *d_graph = nullptr;
  mask_word_t *d_res_masks = nullptr;
  hiprandState *d_rng_states = nullptr;

  // sizing
  size_t grid_size = 0, block_size = 0, n_blocks = 0;
  size_t warp_step = 0;       // 1: per-thread, warp-size: per-warp
  size_t max_batch_size = 0;  // walks per batch
  size_t mask_words = 0;
} cuda_conf;

__global__ void kernel_rng_setup(hiprandState *d_rng_states,
                                 unsigned long long seed, size_t warp_size) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid % warp_size == 0) {
    int wid = tid / warp_size;
    hiprand_init(seed, wid, 0, d_rng_states + wid);
  }
}

#if CUDA_PROFILE
enum breakdown_tag { OVERALL, KERNEL, COPY, TRANSLATE_ALLOC, TRANSLATE_BUILD };
std::unordered_map<breakdown_tag, std::vector<std::chrono::nanoseconds>>
    profile_breakdown;

void print_profile(breakdown_tag tag, const std::string &label) {
  std::sort(profile_breakdown[tag].begin(), profile_breakdown[tag].end());
  auto &sample(profile_breakdown[tag]);
  std::chrono::microseconds tot{0};
  for (auto &x : sample)
    tot += std::chrono::duration_cast<std::chrono::microseconds>(x);
  std::cout << "*** tag: " << label << "\n*** "
            << "cnt=" << sample.size() << "\tmin=" << sample[0].count()
            << "\tmed=" << sample[sample.size() / 2].count()
            << "\tmax=" << sample.back().count() << "\ttot(us)=" << tot.count()
            << std::endl;
}
#endif

void cuda_init(const cuda_GraphTy &G, unsigned long long seed,
               im::linear_threshold_tag &&model_tag) {
  hipError_t e;
  hipGetDeviceProperties(&cuda_conf.cuda_prop, 0);
  // copy graph to device
  cuda_conf.graph = &G;
  cuda_conf.d_graph = make_cuda_graph(G);

  // sizing
  // cuda_conf.warp_step = 1;  // per thread
  cuda_conf.warp_step = cuda_conf.cuda_prop.warpSize;  // per warp
  cuda_conf.block_size = cuda_conf.warp_step * (1 << 0);
  cuda_conf.n_blocks = 1 << 15;
  cuda_conf.grid_size = cuda_conf.n_blocks * cuda_conf.block_size;
  cuda_conf.max_batch_size = cuda_conf.grid_size / cuda_conf.warp_step;
  cuda_conf.mask_words = MAX_SET_SIZE;

  // print sizing info
  CUDA_LOG("> *** CUDA_BATCHED sizing ***\n");
  CUDA_LOG("block-size = %d\n", cuda_conf.block_size);
  CUDA_LOG("n. blocks  = %d\n", cuda_conf.n_blocks);
  CUDA_LOG("warp size  = %d\n", cuda_conf.cuda_prop.warpSize);
  CUDA_LOG("grid size  = %d\n", cuda_conf.grid_size);
  CUDA_LOG("batch size = %d\n", cuda_conf.max_batch_size);
  CUDA_LOG("g-mem size = %d\n",
           cuda_conf.grid_size * cuda_conf.mask_words * sizeof(mask_word_t));

  // allocate host-side memory for result masks
  auto mask_size = cuda_conf.mask_words * sizeof(mask_word_t);
  cuda_conf.res_masks =
      (mask_word_t *)malloc(cuda_conf.max_batch_size * mask_size);

  // allocate device-side memory for results masks
  e = hipMalloc(&cuda_conf.d_res_masks, cuda_conf.max_batch_size * mask_size);
  cuda_check(e, __FILE__, __LINE__);

  // init rng
  hipMalloc(&cuda_conf.d_rng_states,
             cuda_conf.max_batch_size * sizeof(hiprandState));
  cuda_check(e, __FILE__, __LINE__);

  kernel_rng_setup<<<cuda_conf.n_blocks, cuda_conf.block_size>>>(
      cuda_conf.d_rng_states, seed, cuda_conf.warp_step);
  cuda_check(__FILE__, __LINE__);
}  // namespace im

void cuda_init(const cuda_GraphTy &G, unsigned long long seed,
               im::independent_cascade_tag &&) {}

void cuda_fini(im::linear_threshold_tag &&) {
// print profiling
#if CUDA_PROFILE
  printf("*** profiling summary (time unit: ns) ***\n");

  // print sizing info
  printf("> *** CUDA_BATCHED sizing ***\n");
  printf("block-size = %d\n", cuda_conf.block_size);
  printf("n. blocks  = %d\n", cuda_conf.n_blocks);
  printf("warp size  = %d\n", cuda_conf.cuda_prop.warpSize);
  printf("grid size  = %d\n", cuda_conf.grid_size);
  printf("batch size = %d\n", cuda_conf.max_batch_size);
  printf("g-mem size = %d\n",
         cuda_conf.grid_size * cuda_conf.mask_words * sizeof(mask_word_t));

  print_profile(breakdown_tag::OVERALL, "overall");
  print_profile(breakdown_tag::KERNEL, "kernel");
  print_profile(breakdown_tag::COPY, "device-to-host copy");
  print_profile(breakdown_tag::TRANSLATE_BUILD, "translate > build");
  print_profile(breakdown_tag::TRANSLATE_ALLOC, "translate > build > alloc");
#endif

  // finalize streams and free memory
  assert(cuda_conf.res_masks);
  free(cuda_conf.res_masks);
  assert(cuda_conf.d_res_masks);
  hipFree(cuda_conf.d_res_masks);
  assert(cuda_conf.d_rng_states);
  hipFree(cuda_conf.d_rng_states);

  // cleanup
  destroy_cuda_graph(cuda_conf.d_graph);
}

void cuda_fini(im::independent_cascade_tag &&) {}

template <typename HostGraphTy>
__global__ void kernel_lt_per_thread(
    size_t bs, typename HostGraphTy::DestinationTy **index, size_t num_nodes,
    size_t warp_size, hiprandState *d_rng_states, mask_word_t *d_res_masks) {
  using destination_type = typename HostGraphTy::DestinationTy;
  using vertex_type = typename HostGraphTy::vertex_type;

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid % warp_size == 0) {
    int wid = tid / warp_size;
    if (wid < bs) {
      size_t res_size = 0;

      // init res memory
      auto d_res_mask = d_res_masks + wid * MAX_SET_SIZE;
      memset(d_res_mask, 0, MAX_SET_SIZE * sizeof(mask_word_t));

      // cache rng state
      auto rng_state = d_rng_states + wid;

      // select source node
      vertex_type src = hiprand(rng_state) % num_nodes;
      d_res_mask[res_size++] = src;

      float threshold;
      destination_type *first, *last;
      vertex_type v;
      while (src != num_nodes) {
        // rng
        threshold = hiprand_uniform(rng_state);

        // scan neighbor list
        first = index[src];
        last = index[src + 1];
        src = num_nodes;
        for (; first != last; ++first) {
          threshold -= first->weight;
          if (threshold <= 0) {
            // found candidate vertex
            v = first->vertex;

            // insert if not visited
            size_t i = 0;
            while (i < res_size && d_res_mask[i] != v) ++i;
            if (i == res_size) {
              src = v;
              d_res_mask[res_size++] = v;
            }
            break;
          }
        }
      }

      // mark end-of-set
      if (res_size < MAX_SET_SIZE) d_res_mask[res_size] = num_nodes;
    }  // end if active warp
  }    // end if active thread-in-warp
}  // namespace im

void batch_kernel(size_t batch_size) {
  CUDA_LOG("> [batch_kernel] size=%d\n", batch_size);

#if CUDA_PROFILE
  auto start = std::chrono::high_resolution_clock::now();
#endif

  kernel_lt_per_thread<cuda_GraphTy>
      <<<cuda_conf.n_blocks, cuda_conf.block_size>>>(
          batch_size, cuda_conf.d_graph->d_index_, cuda_conf.graph->num_nodes(),
          cuda_conf.warp_step, cuda_conf.d_rng_states, cuda_conf.d_res_masks);
  cuda_check(__FILE__, __LINE__);

#if CUDA_PROFILE
  // un-comment the following line to measure effective kernel run-time (rather
  // than launch-time)
  // hipDeviceSynchronize();
  auto elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(
      std::chrono::high_resolution_clock::now() - start);
  profile_breakdown[breakdown_tag::KERNEL].push_back(elapsed);
#endif
}

void batch_d2h(size_t batch_size) {
  CUDA_LOG("> [batch_d2h] size=%d\n", batch_size);

#if CUDA_PROFILE
  auto start = std::chrono::high_resolution_clock::now();
#endif
  hipMemcpy(cuda_conf.res_masks, cuda_conf.d_res_masks,
             batch_size * cuda_conf.mask_words * sizeof(mask_word_t),
             hipMemcpyDeviceToHost);

#if CUDA_PROFILE
  auto elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(
      std::chrono::high_resolution_clock::now() - start);
  profile_breakdown[breakdown_tag::COPY].push_back(elapsed);
#endif
}

void batch_build(cuda_res_t &rrr_sets, size_t bf, size_t batch_size) {
  // translate
  CUDA_LOG("> [batch_build] size=%d first=%d\n", batch_size, bf);
#if CUDA_PROFILE
  auto start = std::chrono::high_resolution_clock::now();
  std::chrono::nanoseconds m_elapsed{0};
#endif
  for (size_t i = 0; i < batch_size; ++i) {
    auto &rrr_set = rrr_sets[bf + i];
#if CUDA_PROFILE
    auto m_start = std::chrono::high_resolution_clock::now();
#endif
    rrr_set.reserve(MAX_SET_SIZE);
#if CUDA_PROFILE
    m_elapsed += std::chrono::duration_cast<std::chrono::nanoseconds>(
        std::chrono::high_resolution_clock::now() - m_start);
#endif
    auto res_mask = cuda_conf.res_masks + (i * cuda_conf.mask_words);
    for (size_t j = 0; j < cuda_conf.mask_words &&
                       res_mask[j] != cuda_conf.graph->num_nodes();
         ++j) {
      rrr_set.push_back(res_mask[j]);
    }

#if CUDA_CHECK
    check_lt(rrr_set, *cuda_conf.graph, bf + i);
#endif

    if (rrr_set.size() == MAX_SET_SIZE) {
      fprintf(stderr, "> an RRR set hit the maximum size %d\n", MAX_SET_SIZE);
      exit(1);
    }
  }
#if CUDA_PROFILE
  auto elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(
      std::chrono::high_resolution_clock::now() - start);
  profile_breakdown[breakdown_tag::TRANSLATE_BUILD].push_back(elapsed);
  profile_breakdown[breakdown_tag::TRANSLATE_ALLOC].push_back(m_elapsed);
#endif
}

cuda_res_t CudaGenerateRRRSets(size_t theta,
                               im::linear_threshold_tag &&model_tag) {
  CUDA_LOG("> *** CudaGenerateRRRSets theta=%d ***\n", theta);

#if CUDA_PROFILE
  auto start = std::chrono::high_resolution_clock::now();
#endif

  cuda_res_t rrr_sets(theta);

  auto remainder = rrr_sets.size();
  size_t batch_first = 0;

  while (remainder) {
    auto batch_size = std::min(remainder, cuda_conf.max_batch_size);

    batch_kernel(batch_size);
    batch_d2h(batch_size);
    batch_build(rrr_sets, batch_first, batch_size);

    // build sets for batch i
    remainder -= batch_size;
    batch_first += batch_size;
  }

#if CUDA_PROFILE
  auto elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(
      std::chrono::high_resolution_clock::now() - start);
  profile_breakdown[breakdown_tag::OVERALL].push_back(elapsed);
#endif

  return rrr_sets;
}  // namespace im

cuda_res_t CudaGenerateRRRSets(size_t theta,
                               im::independent_cascade_tag &&model_tag) {
  assert(false);
  return cuda_res_t{};
}

}  // namespace im
