//===------------------------------------------------------------*- C++ -*-===//
//
// Copyright 2018 Battelle Memorial Institute
//
//===----------------------------------------------------------------------===//

#include <cstdio>

#include "ripples/cuda/cuda_utils.h"

namespace ripples {

  //
  // debug utilities
  //
  void cuda_check(hipError_t err, const char *fname, int line) {
    if (err != hipSuccess) {
      fprintf(stderr, "> CUDA error @%s:%d: name=%s msg='%s'\n", fname, line,
                    hipGetErrorName(err), hipGetErrorString(err));
      fflush(stderr);
    }
  }
  
  void cuda_check(const char *fname, int line) {
    cuda_check(hipGetLastError(), fname, line);
  }
  }  // namespace ripples