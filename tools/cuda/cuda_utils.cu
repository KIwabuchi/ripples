//===------------------------------------------------------------*- C++ -*-===//
//
// Copyright 2018 Battelle Memorial Institute
//
//===----------------------------------------------------------------------===//

#include "spdlog/spdlog.h"

#include "ripples/cuda/cuda_utils.h"

namespace ripples {

  //
  // debug utilities
  //
  void cuda_check(hipError_t err, const char *fname, int line) {
    if (err != hipSuccess) {
      spdlog::error("> CUDA error @%s:%d: name=%s msg='%s'\n", fname, line,
                    hipGetErrorName(err), hipGetErrorString(err));
    }
  }
  
  void cuda_check(const char *fname, int line) {
    cuda_check(hipGetLastError(), fname, line);
  }
  }  // namespace ripples